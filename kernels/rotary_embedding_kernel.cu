#include "hip/hip_runtime.h"
#include <stdint.h>

#ifndef USE_ROCM
  #define VLLM_LDG(arg) __ldg(arg)
#else
  #define VLLM_LDG(arg) *(arg)
#endif

template<typename scalar_t, bool IS_NEOX>
inline __device__ void apply_rotary_embedding(
  scalar_t* __restrict__ arr,
  const scalar_t* __restrict__ cos_ptr,
  const scalar_t* __restrict__ sin_ptr,
  int rot_offset,
  int embed_dim)
{
  int x_index, y_index;
  scalar_t cos, sin;
  if (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    x_index = rot_offset;
    y_index = embed_dim + rot_offset;
    cos = VLLM_LDG(cos_ptr + x_index);
    sin = VLLM_LDG(sin_ptr + x_index);
  } else {
    // GPT-J style rotary embedding.
    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    cos = VLLM_LDG(cos_ptr + x_index / 2);
    sin = VLLM_LDG(sin_ptr + x_index / 2);
  }

  const scalar_t x = arr[x_index];
  const scalar_t y = arr[y_index];
  arr[x_index] = x * cos - y * sin;
  arr[y_index] = y * cos + x * sin;
}

template<typename scalar_t, bool IS_NEOX>
__global__ void rotary_embedding_kernel(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  scalar_t* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const scalar_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const scalar_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const scalar_t* cos_ptr = cache_ptr;
  const scalar_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<scalar_t, IS_NEOX>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<scalar_t, IS_NEOX>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

extern "C" __global__ void rotary_embedding_kernel_neox(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  scalar_t* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const scalar_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  rotary_embedding_kernel<scalar_t, true>(positions, query, key, cos_sin_cache, rot_dim, query_stride, key_stride, num_heads, num_kv_heads, head_size);
}

extern "C" __global__ void rotary_embedding_kernel_normal(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  scalar_t* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const scalar_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  rotary_embedding_kernel<scalar_t, true>(positions, query, key, cos_sin_cache, rot_dim, query_stride, key_stride, num_heads, num_kv_heads, head_size);
}
